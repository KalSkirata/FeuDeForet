#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "cuPrintf.cu"

#define N 16

#define TILE_width 16


// for Linux platform, plz make sure the size of data type is correct for BMP spec.
// if you use this on Windows or other platforms, plz pay attention to this.
typedef int LONG;
typedef unsigned char BYTE;
typedef unsigned int DWORD;
typedef unsigned short WORD;

// __attribute__((packed)) on non-Intel arch may cause some unexpected error, plz be informed.

typedef struct tagBITMAPFILEHEADER
{
  WORD    bfType; // 2  /* Magic identifier */
  DWORD   bfSize; // 4  /* File size in bytes */
  WORD    bfReserved1; // 2
  WORD    bfReserved2; // 2
  DWORD   bfOffBits; // 4 /* Offset to image data, bytes */ 
} __attribute__((packed)) BITMAPFILEHEADER;

typedef struct tagBITMAPINFOHEADER
{
  DWORD    biSize; // 4 /* Header size in bytes */
  LONG     biWidth; // 4 /* Width of image */
  LONG     biHeight; // 4 /* Height of image */
  WORD     biPlanes; // 2 /* Number of colour planes */
  WORD     biBitCount; // 2 /* Bits per pixel */
  DWORD    biCompress; // 4 /* Compression type */
  DWORD    biSizeImage; // 4 /* Image size in bytes */
  LONG     biXPelsPerMeter; // 4
  LONG     biYPelsPerMeter; // 4 /* Pixels per meter */
  DWORD    biClrUsed; // 4 /* Number of colours */ 
  DWORD    biClrImportant; // 4 /* Important colours */ 
} __attribute__((packed)) BITMAPINFOHEADER;

/*
  typedef struct tagRGBQUAD
  {
  unsigned char    rgbBlue;   
  unsigned char    rgbGreen;
  unsigned char    rgbRed;  
  unsigned char    rgbReserved;
  } RGBQUAD;
  * for biBitCount is 16/24/32, it may be useless
  */

typedef struct
{
  BYTE    b;
  BYTE    g;
  BYTE    r;
} RGB_data; // RGB TYPE, plz also make sure the order

int bmp_generator(char *filename, int width, int height, unsigned char *data)
{
  BITMAPFILEHEADER bmp_head;
  BITMAPINFOHEADER bmp_info;
  int size = width * height * 3;

  bmp_head.bfType = 0x4D42; // 'BM'
  bmp_head.bfSize= size + sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER); // 24 + head + info no quad    
  bmp_head.bfReserved1 = bmp_head.bfReserved2 = 0;
  bmp_head.bfOffBits = bmp_head.bfSize - size;
  // finish the initial of head

  bmp_info.biSize = 40;
  bmp_info.biWidth = width;
  bmp_info.biHeight = height;
  bmp_info.biPlanes = 1;
  bmp_info.biBitCount = 24; // bit(s) per pixel, 24 is true color
  bmp_info.biCompress = 0;
  bmp_info.biSizeImage = size;
  bmp_info.biXPelsPerMeter = 0;
  bmp_info.biYPelsPerMeter = 0;
  bmp_info.biClrUsed = 0 ;
  bmp_info.biClrImportant = 0;
  // finish the initial of infohead;

  // copy the data
  FILE *fp;
  if (!(fp = fopen(filename,"wb"))) return 0;

  fwrite(&bmp_head, 1, sizeof(BITMAPFILEHEADER), fp);
  fwrite(&bmp_info, 1, sizeof(BITMAPINFOHEADER), fp);
  fwrite(data, 1, size, fp);
  fclose(fp);

  return 1;
}

#define TAB(t, x, y) (t)[(y)*width+(x)]

RGB_data woods;
RGB_data ground;
RGB_data fire;
RGB_data ash;

RGB_data buffer[256][256], *bufferGPU;

int cptFire=0;
int cptWoods=0;

int xGPU=-1;
int yGPU=-1;

void init_color(){
  woods.b=0;
  woods.g=204;
  woods.r=0;
	
  ground.b=0;
  ground.g=51;
  ground.r=102;
	
  fire.b=0;
  fire.g=0;
  fire.r=255;

  ash.b=128;
  ash.g=128;
  ash.r=128;
}


void init_grid(int width, int height, int percentage){
  int  i,j,r;
  srand(time(NULL));
  for (i = 0; i < width; i++)
    {
      for (j = 0; j < height; j++)
	{
	  r=rand()%10;
	  if(r < percentage/10){ //70% de chance que ce soit de la foret
	    buffer[i][j].g = woods.g;
	    buffer[i][j].b = woods.b;
	    buffer[i][j].r = woods.r;
	    cptWoods+=1;
	  }else{
	    buffer[i][j].g = ground.g;
	    buffer[i][j].b = ground.b;
	    buffer[i][j].r = ground.r;
	  }
	}
    }

  //bmp_generator("./test.bmp", width, height, (BYTE*)buffer);

}


void init_fire(int width, int height){

  int x,y;	

  srand(time(NULL));
  x=rand()%width;
  y=rand()%height;
	
  while(buffer[y][x].g==ground.g){
    x=rand()%width;
    y=rand()%height;
  }
  printf("init_fire : x=%d y=%d \n",x,y);

  buffer[y][x].r=fire.r;
  buffer[y][x].g=fire.g;
  buffer[y][x].b=fire.b;
  bmp_generator("./test.bmp", width, height, (BYTE*)buffer);

} 


void put_fireCPU(int width, int height){
	
  int x,y,left,right,down,up;
	
  for(y=0; y<height; y++){
    for(x=0; x<width; x++){	
      if (buffer[y][x].g==woods.g){
	//Contrôle dépassement tableau
	if(x<width-1){
	  right=x+1;
	}
			
	if(x>0){
	  left=x-1;
	}else{
	  left=0;			
	}
		
	if(y<height-1){
	  up=y+1;
	}

	if(y>0){
	  down=y-1;
	}else{
	  down=0;			
	}

	//On regarde les 4 cases à côté
	if( (buffer[up][x].r==fire.r) || (buffer[down][x].r==fire.r) || (buffer[y][right].r==fire.r) || (buffer[y][left].r==fire.r) ){

	  buffer[y][x].r=fire.r;
	  buffer[y][x].g=fire.g;
	  buffer[y][x].b=fire.b;
	  cptFire+=1;
	}
      }
    }
  }
}



__global__ void putFireGPU(RGB_data woods, RGB_data fire, RGB_data ash, int *cptFire,RGB_data *bufferGPU, int width, int height){
	
  uint x,y;
  int left,right,down,up;

  x=(blockIdx.x * blockDim.x) + threadIdx.x;
  y=(blockIdx.y * blockDim.y) + threadIdx.y;

  if (x >= width || y >= height) return;

  if (TAB(bufferGPU, x, y).g==woods.g){
    //Contrôle dépassement tableau
    if(x<width-1)
      right=x+1;
    else right = width-1;

    if(x>0){
      left=x-1;
    }else{
      left=0;			
    }
	  
    if(y<height-1)
      up=y+1;
    else up = height-1;
	  
    if(y>0){
      down=y-1;
    }else{
      down=0;			
    }

    //TODO improve

    if(TAB(bufferGPU, left, y).r==fire.r){
      //put current position in fire
      TAB(bufferGPU, x, y).r=fire.r;
      TAB(bufferGPU, x, y).g=fire.g;
      TAB(bufferGPU, x, y).b=fire.b;
      //TAB(cptFire,0,0)++;
      //put neighbour in ash
      TAB(bufferGPU, left, y).r=ash.r;
      TAB(bufferGPU, left, y).g=ash.g;
      TAB(bufferGPU, left, y).b=ash.b;
    }else if(TAB(bufferGPU, right, y).r==fire.r){      
      //put current position in fire
      TAB(bufferGPU, x, y).r=fire.r;
      TAB(bufferGPU, x, y).g=fire.g;
      TAB(bufferGPU, x, y).b=fire.b;
      //TAB(cptFire,0,0)++;
      //put neighbour in ash
      TAB(bufferGPU, right, y).r=ash.r;
      TAB(bufferGPU, right, y).g=ash.g;
      TAB(bufferGPU, right, y).b=ash.b;
    }else if(TAB(bufferGPU, x, up).r==fire.r){   
      //put current position in fire
      TAB(bufferGPU, x, y).r=fire.r;
      TAB(bufferGPU, x, y).g=fire.g;
      TAB(bufferGPU, x, y).b=fire.b;
      //TAB(cptFire,0,0)++;
      //put neighbour in ash
      TAB(bufferGPU, x, up).r=ash.r;
      TAB(bufferGPU, x, up).g=ash.g;
      TAB(bufferGPU, x, up).b=ash.b;
    }else if(TAB(bufferGPU, x, down).r==fire.r){   
      //put current position in fire
      TAB(bufferGPU, x, y).r=fire.r;
      TAB(bufferGPU, x, y).g=fire.g;
      TAB(bufferGPU, x, y).b=fire.b;
      //TAB(cptFire,0,0)++;
      //put neighbour in ash
      TAB(bufferGPU, x, down).r=ash.r;
      TAB(bufferGPU, x, down).g=ash.g;
      TAB(bufferGPU, x, down).b=ash.b;
    }
  }
}


int main(int argc, char **argv)
{

  //dim3 dimGrid(ceil((float)width/TILE_width), ceil((float)height/TILE_width));
  dim3 dimBlock(TILE_width, TILE_width, 1);
  int i;
  char test[16];

  double pourcentageFeu;

  while(argc < 3){
    printf("Usage : ./projet PERCENTAGE_WOODS PERCENTAGE_STOP \n - PERCENTAGE_WOODS : Percentage of woods at the initialization (0-100%)\n - PERCENTAGE_STOP : Percentage of fire to stop the spread \n");
    return -1;
  }

  int width = 256;
  int height = 256;
  int percentage_woods = atoi(argv[1]);
  int percentage_stop = atoi(argv[2]);
  
  dim3 dimGrid(ceil((float)width/TILE_width), ceil((float)height/TILE_width));

  //Initialisation du buffer (de l'image)
  memset(buffer, 0, sizeof(RGB_data) * size_t(height*width)); //CPU

  //Initialisation de la grille
  init_color();
  init_grid(width, height, 60);
  init_fire(width, height);

  cudaPrintfInit();

  //GPU
  hipMalloc((void**)&bufferGPU,  sizeof(RGB_data) * size_t(height*width)); 
  hipMemset(bufferGPU, 0,  sizeof(RGB_data) * size_t(height*width));

  hipMemcpy(bufferGPU, buffer, sizeof(RGB_data) * size_t(height*width), hipMemcpyHostToDevice);
  
  int* d_cptFire;
  hipMalloc((void**)&d_cptFire, sizeof(int));
  hipMemset(d_cptFire, 0, sizeof(int));
  i = 0;
  while (i<100){
    //put_fire();
    //printf("%d\n", i);
	  
    putFireGPU <<< dimGrid,dimBlock >>> (woods, fire, ash, d_cptFire, bufferGPU, width, height);
    cudaPrintfDisplay(stdout, true);
    hipMemcpy(buffer,bufferGPU, sizeof(RGB_data) * size_t(height*width),hipMemcpyDeviceToHost); 

    sprintf(test, "test%03d.bmp", i);
    i++;

    bmp_generator(test, width, height, (BYTE*)buffer);

    //int *h_cptFire;
    //hipMemcpy(h_cptFire, d_cptFire, sizeof(int),hipMemcpyDeviceToHost);
    //pourcentageFeu=((double)*h_cptFire/(double)width*height)*100.0;
    //printf("Poucentage feu: %f - cptFire=%d\n", pourcentageFeu, h_cptFire);
  }

  cudaPrintfEnd();
  printf("Propagation finie \n");

  hipFree(bufferGPU);
  hipFree(d_cptFire);

  return 0;
}
